#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

const char* comp = "comp";
const char* comp_large = "comp_large";
const char* merge = "merge";
const char* comp_small = "comp_small";
const char* sequential_sort = "sequential_sort";
const char* whole_computation = "whole_computation";
const char* data_init = "data_init";
const char* transfer_host_to_device = "transfer_host_to_device";
const char* transfer_device_to_host = "transfer_device_to_host";
const char* correctness_check = "correctness_check";


int NUMTHREADS;
int sizeOfMatrix;
int BLOCKS;

__device__ void sequential_sort(int* data, int size) {
    for (int i = 1; i < size; i++) {
        int key = data[i];
        int j = i - 1;
        while (j >= 0 && data[j] > key) {
            data[j + 1] = data[j];
            j = j - 1;
        }
        data[j + 1] = key;
    }
}

__device__ void merge(int* data, int start, int middle, int end) {
    int i = start;
    int j = middle;
    int* tempData = data;
    for(int k = start; k < end; k++) {
        if(i < middle && (j>=end || data[i] < data[j])) {
            data[k] = tempData[j];
            i++;
        }
        else {
            data[k] = tempData[i];
            j++;
        }
    }
    hipFree(tempData);
}

__device__ void recursive_merge(double *data, int size, int aggregate) {
    int taskid = threadIdx.x;
    if(taskid%aggregate == 1) {
        //sequential merge with start = taskid * (size / blockDim.x), mid = (taskid + 1) * (size / blockDim.x), end = (taskid + (aggregate/2)) * (size / blockDim.x)
        int start = taskid * (size / blockDim.x);
        int mid = (taskid + 1) * (size / blockDim.x);
        int end = (taskid + (aggregate/2)) * (size / blockDim.x);
        CALI_MARK_BEGIN(comp);
        CALI_MARK_BEGIN(comp_large);
        CALI_MARK_BEGIN(merge);
        merge(data, start, mid, end);
        CALI_MARK_END(comp);
        CALI_MARK_END(comp_large);
        CALI_MARK_END(merge);
        
        //recursively merge up
        if((taskid + (aggregate/2)) * (size / blockDim.x) != size) {
            recursive_merge(data, size, aggregate*2);
        }
    }
}

// Define the CUDA kernel for performing merge sort on each block
__global__ void mergeSortKernel(double *data, int size) {
    //figure out which section of the shared, block-level data we're working with
    CALI_CXX_MARK_FUNCTION;
    int taskid = threadIdx.x;
    int start = taskid * (size / blockDim.x);
    int end = (taskid + 1) * (size / blockDim.x);

    //sort data in our little subsection
    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_small);
    CALI_MARK_BEGIN(sequential_sort);
    sequential_sort(data[start], end-start);
    CALI_MARK_END(comp);
    CALI_MARK_END(comp_small);
    CALI_MARK_END(sequential_sort);

    //recursive merge up
    recursive_merge(data, size, 2);
}

// Define the CUDA kernel for merging sorted blocks, should be performed on a single block theoretically with threads = num_blocks
__global__ void mergeKernel(double *data, int size) {
    //recursive merge up
    recursive_merge(data, size, 2);
}

bool test_array_is_in_order(int arr[]) {
    for (int i = 1; i < LENGTH; i ++) {
        if (arr[i] < arr[i - 1]) {
            printf("Error. Out of order sequence: %d found\n", arr[i]);
            return false;
        }
    }
    printf("Array is in sorted order\n");
    return true;
};

int main() {
    CALI_CXX_MARK_FUNCTION;

    CALI_MARK_BEGIN(whole_computation);

    cali::ConfigManager mgr;
    mgr.start();
 
    NUMTHREADS = atoi(argv[1]);
    sizeOfMatrix = atoi(argv[2]);
    BLOCKS = sizeOfMatrix / NUMTHREADS;

    double[] localArr = new double[sizeOfMatrix];
    double* remoteArr;
    hipMalloc((void**)&remoteArr, sizeOfMatrix);

    CALI_MARK_BEGIN(data_init);
    for(int i = 0; i < sizeOfMatrix; i++) {
        localArr[i] = srand(1);
    }
    CALI_MARK_END(data_init);

    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);
    CALI_MARK_BEGIN(transfer_host_to_device);
    hipMemcpy(remoteArr, localArr, sizeOfMatrix, hipMemcpyHostToDevice);
    CALI_MARK_END(comm);
    CALI_MARK_END(comm_large);
    CALI_MARK_END(transfer_host_to_device);

    dim3 blocks(BLOCKS-1,1);    /* Number of blocks   */
    dim3 threads(NUMTHREADS,1);  /* Number of threads  */
    dim3 kernelBlock(1,1); 

    // Launch the CUDA kernel for performing merge sort on each block
    mergeSortKernel<<<blocks,threads>>>(remoteArr,sizeOfMatrix);

    // Launch the CUDA kernel for merging the sorted blocks
    mergeKernel<<<kernelBlock,threads>>>(remoteArr,sizeOfMatrix);

    // Copy the sorted data back from the GPU to the host
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);
    CALI_MARK_BEGIN(transfer_device_to_host);
    hipMemcpy(localArr, remoteArr, sizeOfMatrix, hipMemcpyDeviceToHost);
    CALI_MARK_END(comm);
    CALI_MARK_END(comm_large);
    CALI_MARK_END(transfer_device_to_host);

    // Free memory on the GPU
    hipFree(remoteArr);

    // Print the sorted data
    CALI_MARK_BEGIN(correctness_check)
    test_array_is_in_order(localArr);
    CALI_MARK_END(correctness_check)

    CALI_MARK_END(whole_computation);
    

    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", "MergeSort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "CUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "Double"); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", sizeof(Double)); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", sizeOfMatrix); // The number of elements in input dataset (1000)
    adiak::value("InputType", "Random"); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    adiak::value("num_procs", num_tasks); // The number of processors (MPI ranks)
    adiak::value("num_threads", threads); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", blocks+kernelBlock); // The number of CUDA blocks 
    adiak::value("group_num", 25); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "Online/AI/Handwritten") // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").

    return 0;
}
