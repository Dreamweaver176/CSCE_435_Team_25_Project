#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <iostream>
#include <algorithm>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>
#include <thrust/sort.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int THREADS;
int BLOCKS;
int NUM_VALS;

/* Define Caliper region names */
const char* whole_computation = "whole_computation";
const char* data_init = "data_init";
const char* comm = "comm";
const char* comm_small = "comm_small";
const char* comm_large = "comm_large";
const char* comp = "comp";
const char* comp_small = "comp_small";
const char* comp_large = "comp_large";
const char* correctness_check = "correctness_check";

void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.3fs\n", elapsed);
};

void array_print(int *arr, int length) 
{
  int i;
  for (i = 0; i < length; ++i) {
    printf("%1.3f ",  arr[i]);
  }
  printf("\n");
};

void array_fill(int *arr, int length)
{
    // for (int i = 0; i < length; ++i) {               // Sorted
    //     arr[i] = static_cast<int>(i);
    // }

    srand(time(NULL));                               // Random
    int i;
    for (i = 0; i < length; ++i) {
        arr[i] = static_cast<int>(rand() % length);
    }

    // for (int i = 0; i < length; ++i) {               // Reverse sorted
    //     arr[i] = static_cast<int>(length - i);
    // }

    // srand(time(NULL));                             // 1%perturbed
    // for(int i = 0; i < length; i++) {
    //     arr[i] = static_cast<int>(i);
    //     if (rand() % 100 == 1) {
    //         arr[i] *= static_cast<int>(rand() % 10 + 0.5);
    //     }
    // }
};

__global__ void merge(int* values, int* temp, int NUM_VALS) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int left = 2 * idx * NUM_VALS / (2 * blockDim.x * gridDim.x);
    int right = (2 * idx + 1) * NUM_VALS / (2 * blockDim.x * gridDim.x);
    int end = (2 * idx + 2) * NUM_VALS / (2 * blockDim.x * gridDim.x);
    
    if (end > NUM_VALS) end = NUM_VALS;

    int i = left;
    int j = right;
    int k = 0;

    while (i < right && j < end) {
        if (values[i] <= values[j]) {
            temp[k++] = values[i++];
        } else {
            temp[k++] = values[j++];
        }
    }

    while (i < right) {
        temp[k++] = values[i++];
    }

    while (j < end) {
        temp[k++] = values[j++];
    }

    // Copy the merged data back to the original array
    for (int i = left, m = 0; i < end; ++i, ++m) {
        values[i] = temp[m];
    }
};

void sample_sort(int* values) {
    CALI_CXX_MARK_FUNCTION;

    int block_size = THREADS;
    int grid_size = BLOCKS;

    int* d_values;

    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);

    hipMalloc((void**)&d_values, NUM_VALS * sizeof(int));
    hipMemcpy(d_values, values, NUM_VALS * sizeof(int), hipMemcpyHostToDevice);

    CALI_MARK_END(comm_large);
    CALI_MARK_END(comm);

    dim3 blocks(BLOCKS,1);    /* Number of blocks   */
    dim3 threads(THREADS,1);  /* Number of threads  */

    int* d_temp;
    hipMalloc((void**)&d_temp, NUM_VALS * sizeof(int));

    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_large);

    // Sort each block independently using CUDA kernel
    for (int i = 0; i < grid_size; ++i) {
        int offset = i * NUM_VALS / grid_size;
        int size = NUM_VALS / grid_size;
        thrust::sort(thrust::device, d_values + offset, d_values + offset + size);
    }

    CALI_MARK_END(comp_large);
    CALI_MARK_END(comp);

    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_small);

    // Merge sorted blocks iteratively until a fully sorted array is obtained
    while (grid_size > 1) {
        int new_grid_size = (grid_size + 1) / 2;
        merge<<<new_grid_size, block_size>>>(d_values, d_temp, NUM_VALS);
        hipDeviceSynchronize();
        grid_size = new_grid_size;
    }

    CALI_MARK_END(comp_small);
    CALI_MARK_END(comp);

    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_small);

    // Copy the sorted array back to the host
    hipMemcpy(values, d_values, NUM_VALS * sizeof(int), hipMemcpyDeviceToHost);

    CALI_MARK_END(comm_small);
    CALI_MARK_END(comm);

    hipFree(d_values);
    hipFree(d_temp);
};

int main(int argc, char *argv[])
{
    CALI_CXX_MARK_FUNCTION;

    THREADS = atoi(argv[1]);
    NUM_VALS = atoi(argv[2]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n", BLOCKS);

    CALI_MARK_BEGIN(whole_computation);

    // Create caliper ConfigManager object
    cali::ConfigManager mgr;
    mgr.start();

    CALI_MARK_BEGIN(data_init);

    int *values = (int*) malloc( NUM_VALS * sizeof(int));
    array_fill(values, NUM_VALS);

    CALI_MARK_END(data_init);

    sample_sort(values);

    CALI_MARK_BEGIN(correctness_check);

    bool sorted = true;
    for (int i = 1; i < NUM_VALS; i++) {
      //printf("a[i]: %d\n", values[i]);
        if (values[i] < values[i-1]) {
            printf("Error. Out of order sequence: %d found\n", values[i]);
            sorted = false;
        }
    }
    if (sorted) {
        printf("Array is in sorted order\n");
    }

    CALI_MARK_END(correctness_check);
    CALI_MARK_END(whole_computation);

    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", "SampleSort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "CUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "Integer"); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", sizeof(int)); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", NUM_VALS); // The number of elements in input dataset (1000)
    adiak::value("InputType", "Random"); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    //adiak::value("num_procs", numtasks); // The number of processors (MPI ranks)
    adiak::value("num_threads", THREADS); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", BLOCKS); // The number of CUDA blocks 
    adiak::value("group_num", 25); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "AI/Handwritten"); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").

    // Flush Caliper output before finalizing MPI
    mgr.stop();
    mgr.flush();

    //free(values);
};
